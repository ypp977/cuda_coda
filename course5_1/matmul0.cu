#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cmath>   // for fabsf
#include <fstream> // for CSV output
#include <iostream>
#include <vector>

#define BLOCK_SIZE 32
#define TOL 1e-5f

void checkCudaError(hipError_t err, const char* msg)
{
    if (err != hipSuccess)
    {
        std::cerr << msg << " CUDA ERROR: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t status, const char* msg)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << msg << " CUBLAS ERROR: " << status << std::endl;
        exit(EXIT_FAILURE);
    }
}

// 手写的SGEMM kernel
__global__ void mysgemm_v0(int M, int N, int K, float alpha, float* A, float* B, float beta,
                           float* C)
{
    // 列
    int gx = blockIdx.x * blockDim.x + threadIdx.x;
    // 行
    int gy = blockIdx.y * blockDim.y + threadIdx.y;

    if (gx < N && gy < N)
    {
        return;
    }

    float tmp = 0.0f;
    for (int i = 0; i < K; i++)
    {
        tmp += A[gy * K + i] * B[i * N + gx];
    }
}

int main()
{
    std::vector<int> sizes = {128, 256, 512, 1024, 2048, 4096, 8192};

    // 打开CSV文件
    std::ofstream csv_file("sgemm_benchmark_v1.csv");
    csv_file << "Size,CUBLAS_GFLOPS,MySGEMM_FLOPS,Matched" << std::endl;

    for (int N : sizes)
    {
        std::cout << "Testing size: " << N << std::endl;

        size_t size = N * N * sizeof(float);
        float* A = (float*)malloc(size);
        float* B = (float*)malloc(size);
        float* C_cublas = (float*)malloc(size);
        float* C_v1 = (float*)malloc(size);

        float *d_A, *d_B, *d_C_v1;
        checkCudaError(hipMalloc(&d_A, size), "hipMalloc d_A failed");
        checkCudaError(hipMalloc(&d_B, size), "hipMalloc d_B failed");
        checkCudaError(hipMalloc(&d_C_v1, size), "hipMalloc d_C_v1 failed");

        bool out_of_memory = false;

        try
        {
            // 初始化矩阵 A 和 B
            for (int i = 0; i < N * N; ++i)
            {
                A[i] = 1.0f;
                B[i] = 2.0f;
            }

            // 拷贝到设备
            checkCudaError(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice),
                           "hipMemcpy A to device failed");
            checkCudaError(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice),
                           "hipMemcpy B to device failed");

            hipblasHandle_t handle;
            checkCublasError(hipblasCreate(&handle), "hipblasCreate failed");

            float alpha = 1.0f;
            float beta = 0.0f;

            hipEvent_t start, stop;
            checkCudaError(hipEventCreate(&start), "hipEventCreate(start) failed");
            checkCudaError(hipEventCreate(&stop), "hipEventCreate(stop) failed");

            // warmup
            int warpup_time = 10; // 热身次数
            for (int i = 0; i < warpup_time; ++i)
            {
                checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B,
                                             N, d_A, N, &beta, d_C_v1, N),
                                 "hipblasSgemm failed");
            }
            hipDeviceSynchronize();

            // cuBLAS SGEMM
            int repeat_time = 5;
            checkCudaError(hipEventRecord(start), "hipEventRecord(start cublas) failed");
            for (int i = 0; i < repeat_time; ++i)
            {
                checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B,
                                             N, d_A, N, &beta, d_C_v1, N),
                                 "hipblasSgemm failed");
            }

            checkCudaError(hipEventRecord(stop), "hipEventRecord(stop cublas) failed");
            checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize cublas failed");

            float cublas_time = 0;
            checkCudaError(hipEventElapsedTime(&cublas_time, start, stop),
                           "hipEventElapsedTime cublas failed");

            // 拷贝 cuBLAS 结果
            checkCudaError(hipMemcpy(C_cublas, d_C_v1, size, hipMemcpyDeviceToHost),
                           "hipMemcpy C_cublas failed");

            // mysgemm_v1
            checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

            dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
            dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

            for (int i = 0; i < warpup_time; ++i)
            {
                mysgemm_v1<<<blocks, threads>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
            }
            hipDeviceSynchronize();

            checkCudaError(hipEventRecord(start), "hipEventRecord(start v1) failed");
            for (int i = 0; i < repeat_time; ++i)
            {
                mysgemm_v1<<<blocks, threads>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
            }
            checkCudaError(hipEventRecord(stop), "hipEventRecord(stop v1) failed");
            checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize v1 failed");

            float v1_time = 0;
            checkCudaError(hipEventElapsedTime(&v1_time, start, stop),
                           "hipEventElapsedTime v1 failed");

            // 拷贝手写 kernel 结果
            checkCudaError(hipMemcpy(C_v1, d_C_v1, size, hipMemcpyDeviceToHost),
                           "hipMemcpy C_v1 failed");
            // 结果比较
            int error_count = 0;
            for (int i = 0; i < N * N && error_count < 10; ++i)
            {
                if (fabsf(C_cublas[i] - C_v1[i]) > TOL)
                {
                    error_count++;
                }
            }

            float cublas_gflops = repeat_time * 2.0f * N * N * N / (cublas_time * 1e6f); // GFlops
            float v1_gflops = repeat_time * 2.0f * N * N * N / (v1_time * 1e6f);         // GFlops
            // 写入CSV
            csv_file << N << "," << cublas_gflops << "," << v1_gflops << ","
                     << (error_count == 0 ? "1" : "0") << std::endl;

            // 释放资源
            hipblasDestroy(handle);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C_v1);

            free(A);
            free(B);
            free(C_cublas);
            free(C_v1);
        }
        catch (...)
        {
            std::cerr << "Out of memory or error during testing size: " << N << std::endl;
            out_of_memory = true;
        }

        if (!out_of_memory)
        {
            std::cout << "Finished size: " << N << std::endl;
        }
        else
        {
            csv_file << N << ",OOM,OOM,0" << std::endl;
        }
    }

    csv_file.close();

    std::cout << "Benchmark completed. Results saved to 'sgemm_benchmark.csv'" << std::endl;
    return 0;
}
