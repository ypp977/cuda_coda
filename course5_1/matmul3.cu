#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cmath>    // for fabsf
#include <fstream>  // for CSV output
#include <iostream>
#include <vector>

#define TOL 1e-5f
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << msg << " CUDA ERROR: " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

void checkCublasError(hipblasStatus_t status, const char *msg) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << msg << " CUBLAS ERROR: " << status << std::endl;
    exit(EXIT_FAILURE);
  }
}
template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void mysgemm_v6(int M, int N, int K, float alpha, float *A, float *B,
                           float beta, float *C) {
  int bx = blockIdx.x;
  int by = blockIdx.y;

  const int block_row_thread = BN / TN;
  const int block_col_thread = BM / TM;
  const int thread_num = block_row_thread * block_col_thread;

  int tx = (threadIdx.x % block_row_thread) * TN;
  int ty = (threadIdx.x / block_row_thread) * TM;

  __shared__ float As[BK * BM];
  __shared__ float Bs[BK * BN];

  const int ldg_a_num = BK * BM / thread_num / 4;
  const int ldg_b_num = BK * BN / thread_num / 4;

  int a_tile_row = threadIdx.x / (BK / 4);
  int a_tile_col = threadIdx.x % (BK / 4) * 4;
  int a_tile_stride = BM / ldg_a_num;

  int b_tile_row = threadIdx.x / (BN / 4);
  int b_tile_col = threadIdx.x % (BN / 4) * 4;
  int b_tile_stride = BK / ldg_b_num;

  float accum[TM][TN] = {0.};

  float ldg_a_reg[4 * ldg_a_num] = {0.};

  float a_frag[TM];
  float b_frag[TN];

  A = &A[by * BM * K];
  B = &B[bx * BN];
  C = &C[by * BM * N + bx * BN];

#pragma unroll
  for (int k = 0; k < K; k += BK) {
#pragma unroll
    for (int i = 0; i < BM; i += a_tile_stride) {
      int ldg_index = i / a_tile_stride * 4;
      FETCH_FLOAT4(ldg_a_reg[ldg_index]) =
          FETCH_FLOAT4(A[OFFSET(a_tile_row + i, a_tile_col, K)]);
      As[OFFSET(a_tile_col, i + a_tile_row, BM)] = ldg_a_reg[ldg_index];
      As[OFFSET(a_tile_col + 1, i + a_tile_row, BM)] = ldg_a_reg[ldg_index + 1];
      As[OFFSET(a_tile_col + 2, i + a_tile_row, BM)] = ldg_a_reg[ldg_index + 2];
      As[OFFSET(a_tile_col + 3, i + a_tile_row, BM)] = ldg_a_reg[ldg_index + 3];
    }
#pragma unroll
    for (int i = 0; i < BK; i += b_tile_stride) {
      FETCH_FLOAT4(Bs[OFFSET(b_tile_row + i, b_tile_col, BN)]) =
          FETCH_FLOAT4(B[OFFSET(b_tile_row + i, b_tile_col, N)]);
    }
    __syncthreads();
    A += BK;
    B += BK * N;
#pragma unroll
    for (int i = 0; i < BK; i++) {
#pragma unroll
      for (int m = 0; m < TM; m += 4) {
        FETCH_FLOAT4(a_frag[m]) = FETCH_FLOAT4(As[OFFSET(i, ty + m, BM)]);
      }
#pragma unroll
      for (int n = 0; n < TN; n += 4) {
        FETCH_FLOAT4(b_frag[n]) = FETCH_FLOAT4(Bs[OFFSET(i, tx + n, BN)]);
      }
#pragma unroll
      for (int m = 0; m < TM; m++) {
#pragma unroll
        for (int n = 0; n < TN; n++) {
          accum[m][n] += a_frag[m] * b_frag[n];
        }
      }
    }
    __syncthreads();
  }
#pragma unroll
  for (int m = 0; m < TM; m++) {
#pragma unroll
    for (int n = 0; n < TN; n += 4) {
      float4 ctmp = FETCH_FLOAT4(C[OFFSET(ty + m, tx + n, N)]);
      ctmp.x = alpha * accum[m][n] + beta * ctmp.x;
      ctmp.y = alpha * accum[m][n + 1] + beta * ctmp.y;
      ctmp.z = alpha * accum[m][n + 2] + beta * ctmp.z;
      ctmp.w = alpha * accum[m][n + 3] + beta * ctmp.w;
      FETCH_FLOAT4(C[OFFSET(ty + m, tx + n, N)]) = ctmp;
    }
  }
}

#define CEIL_DIV(M, N) ((M) + (N) - 1) / (N)
std::vector<int> generateSizes() { return {4096}; }
int main() {
  int device_id = 7;
  checkCudaError(hipSetDevice(device_id), "hipSetDevice failed");
  std::vector<int> sizes = generateSizes();
  // 打开CSV文件
  std::ofstream csv_file("sgemm_benchmark_v4.csv");
  csv_file << "Size,CUBLAS_GFLOPS,MySGEMM_FLOPS,Matched" << std::endl;

  for (int N : sizes) {
    std::cout << "Testing size: " << N << std::endl;

    size_t size = N * N * sizeof(float);
    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C_cublas = (float *)malloc(size);
    float *C_v1 = (float *)malloc(size);

    float *d_A, *d_B, *d_C_v1;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc d_A failed");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc d_B failed");
    checkCudaError(hipMalloc(&d_C_v1, size), "hipMalloc d_C_v1 failed");

    bool out_of_memory = false;

    try {
      // 初始化矩阵 A 和 B
      for (int i = 0; i < N * N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
      }

      // 拷贝到设备
      checkCudaError(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice),
                     "hipMemcpy A to device failed");
      checkCudaError(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice),
                     "hipMemcpy B to device failed");

      hipblasHandle_t handle;
      checkCublasError(hipblasCreate(&handle), "hipblasCreate failed");
      float alpha = 1.0f;
      float beta = 0.0f;

      hipEvent_t start, stop;
      checkCudaError(hipEventCreate(&start), "cudaEventCreate(start) failed");
      checkCudaError(hipEventCreate(&stop), "cudaEventCreate(stop) failed");

      // warmup
      int warpup_time = 10;  // 热身次数
      for (int i = 0; i < warpup_time; ++i) {
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                     &alpha, d_B, N, d_A, N, &beta, d_C_v1, N),
                         "hipblasSgemm failed");
      }
      hipDeviceSynchronize();

      // cuBLAS SGEMM
      int repeat_time = 5;
      checkCudaError(hipEventRecord(start),
                     "cudaEventRecord(start cublas) failed");
      for (int i = 0; i < repeat_time; ++i) {
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                     &alpha, d_B, N, d_A, N, &beta, d_C_v1, N),
                         "hipblasSgemm failed");
      }

      checkCudaError(hipEventRecord(stop),
                     "cudaEventRecord(stop cublas) failed");
      checkCudaError(hipEventSynchronize(stop),
                     "hipEventSynchronize cublas failed");

      float cublas_time = 0;
      checkCudaError(hipEventElapsedTime(&cublas_time, start, stop),
                     "hipEventElapsedTime cublas failed");

      // 拷贝 cuBLAS 结果
      checkCudaError(hipMemcpy(C_cublas, d_C_v1, size, hipMemcpyDeviceToHost),
                     "hipMemcpy C_cublas failed");

      // mysgemm_v1
      checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

      dim3 blockDim(256);
      dim3 gridDim(CEIL_DIV(N, 128), CEIL_DIV(N, 128));

      for (int i = 0; i < warpup_time; ++i) {
        mysgemm_v6<128, 128, 8, 8, 8>
            <<<gridDim, blockDim>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
      }

      hipDeviceSynchronize();
      checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

      checkCudaError(hipEventRecord(start),
                     "cudaEventRecord(start v1) failed");

      for (int i = 0; i < repeat_time; ++i) {
        mysgemm_v6<128, 128, 8, 8, 8>
            <<<gridDim, blockDim>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
      }
      checkCudaError(hipEventRecord(stop), "cudaEventRecord(stop v1) failed");
      checkCudaError(hipEventSynchronize(stop),
                     "hipEventSynchronize v1 failed");
      float v1_time = 0;
      checkCudaError(hipEventElapsedTime(&v1_time, start, stop),
                     "hipEventElapsedTime v1 failed");

      // 拷贝手写 kernel 结果
      checkCudaError(hipMemcpy(C_v1, d_C_v1, size, hipMemcpyDeviceToHost),
                     "hipMemcpy C_v1 failed");
      // 结果比较
      int error_count = 0;
      for (int i = 0; i < N * N && error_count < 10; ++i) {
        if (fabsf(C_cublas[i] - C_v1[i]) > TOL) {
          error_count++;
        }
      }

      float cublas_gflops =
          repeat_time * 2.0f * N * N * N / (cublas_time * 1e6f);  // GFlops
      float v1_gflops =
          repeat_time * 2.0f * N * N * N / (v1_time * 1e6f);  // GFlops
      // 写入CSV
      csv_file << N << "," << cublas_gflops << "," << v1_gflops << ","
               << (error_count == 0 ? "1" : "0") << std::endl;

      // 释放资源
      hipblasDestroy(handle);
      hipEventDestroy(start);
      hipEventDestroy(stop);
      hipFree(d_A);
      hipFree(d_B);
      hipFree(d_C_v1);

      free(A);
      free(B);
      free(C_cublas);
      free(C_v1);

    } catch (...) {
      std::cerr << "Out of memory or error during testing size: " << N
                << std::endl;
      out_of_memory = true;
    }

    if (!out_of_memory) {
      std::cout << "Finished size: " << N << std::endl;
    } else {
      csv_file << N << ",OOM,OOM,0" << std::endl;
    }
  }

  csv_file.close();

  std::cout << "Benchmark completed. Results saved to 'sgemm_benchmark.csv'"
            << std::endl;
  return 0;
}
