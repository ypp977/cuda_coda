#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cmath>    // for fabsf
#include <fstream>  // for CSV output
#include <iostream>
#include <vector>

#define TOL 1e-5f

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << msg << " CUDA ERROR: " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

void checkCublasError(hipblasStatus_t status, const char *msg) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << msg << " CUBLAS ERROR: " << status << std::endl;
    exit(EXIT_FAILURE);
  }
}

template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void mysgemm_v4(int M, int N, int K, float alpha, float *A, float *B,
                           float beta, float *C) {
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int block_row_thread = BN / TN;
  int block_col_thread = BM / TM;
  int thread_num = block_row_thread * block_col_thread;

  int tx = (threadIdx.x % block_row_thread) * TN;
  int ty = (threadIdx.x / block_row_thread) * TM;

  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * BN];

  A = &A[by * BM * K];
  B = &B[bx * BN];
  C = &C[by * BM * N + bx * BN];

  int a_tile_row = threadIdx.x / BK;
  int a_tile_col = threadIdx.x % BK;
  int a_tile_stride = thread_num / BK;

  int b_tile_row = threadIdx.x / BN;
  int b_tile_col = threadIdx.x % BN;
  int b_tile_stride = thread_num / BN;

  float tmp[TM][TN] = {0.};
#pragma unroll
  for (int k = 0; k < K; k += BK) {
#pragma unroll
    for (int i = 0; i < BM; i += a_tile_stride) {
      As[(a_tile_row + i) * BK + a_tile_col] =
          A[(a_tile_row + i) * K + a_tile_col];
    }
#pragma unroll
    for (int i = 0; i < BK; i += b_tile_stride) {
      Bs[(b_tile_row + i) * BN + b_tile_col] =
          B[(b_tile_row + i) * N + b_tile_col];
    }
    __syncthreads();
    A += BK;
    B += BK * N;
#pragma unroll
    for (int i = 0; i < BK; i++) {
#pragma unroll
      for (int j = 0; j < TM; j++) {
        for (int l = 0; l < TN; l++)
          tmp[j][l] += As[(ty + j) * BK + i] * Bs[tx + l + i * BN];
      }
    }
    __syncthreads();
  }
#pragma unroll
  for (int j = 0; j < TM; j++) {
    for (int l = 0; l < TN; l++)
      C[(ty + j) * N + tx + l] =
          alpha * tmp[j][l] + beta * C[(ty + j) * N + tx + l];
  }
}

#define CEIL_DIV(M, N) ((M) + (N) - 1) / (N)

std::vector<int> generateSizes() { return {4096}; }
int main() {
  int device_id = 7;
  checkCudaError(hipSetDevice(device_id), "hipSetDevice failed");
  std::vector<int> sizes = generateSizes();

  // 打开CSV文件
  std::ofstream csv_file("sgemm_benchmark_v3.csv");
  csv_file << "Size,CUBLAS_GFLOPS,MySGEMM_FLOPS,Matched" << std::endl;

  for (int N : sizes) {
    std::cout << "Testing size: " << N << std::endl;

    size_t size = N * N * sizeof(float);
    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C_cublas = (float *)malloc(size);
    float *C_v1 = (float *)malloc(size);

    float *d_A, *d_B, *d_C_v1;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc d_A failed");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc d_B failed");
    checkCudaError(hipMalloc(&d_C_v1, size), "hipMalloc d_C_v1 failed");

    bool out_of_memory = false;

    try {
      // 初始化矩阵 A 和 B
      for (int i = 0; i < N * N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
      }

      // 拷贝到设备
      checkCudaError(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice),
                     "hipMemcpy A to device failed");
      checkCudaError(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice),
                     "hipMemcpy B to device failed");

      hipblasHandle_t handle;
      checkCublasError(hipblasCreate(&handle), "hipblasCreate failed");

      float alpha = 1.0f;
      float beta = 0.0f;

      hipEvent_t start, stop;
      checkCudaError(hipEventCreate(&start), "cudaEventCreate(start) failed");
      checkCudaError(hipEventCreate(&stop), "cudaEventCreate(stop) failed");

      // warmup
      int warpup_time = 10;  // 热身次数
      for (int i = 0; i < warpup_time; ++i) {
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                     &alpha, d_B, N, d_A, N, &beta, d_C_v1, N),
                         "hipblasSgemm failed");
      }
      hipDeviceSynchronize();

      // cuBLAS SGEMM
      int repeat_time = 5;
      checkCudaError(hipEventRecord(start),
                     "cudaEventRecord(start cublas) failed");
      for (int i = 0; i < repeat_time; ++i) {
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                     &alpha, d_B, N, d_A, N, &beta, d_C_v1, N),
                         "hipblasSgemm failed");
      }

      checkCudaError(hipEventRecord(stop),
                     "cudaEventRecord(stop cublas) failed");
      checkCudaError(hipEventSynchronize(stop),
                     "hipEventSynchronize cublas failed");

      float cublas_time = 0;
      checkCudaError(hipEventElapsedTime(&cublas_time, start, stop),
                     "hipEventElapsedTime cublas failed");

      // 拷贝 cuBLAS 结果
      checkCudaError(hipMemcpy(C_cublas, d_C_v1, size, hipMemcpyDeviceToHost),
                     "hipMemcpy C_cublas failed");

      // mysgemm_v4
      checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

      dim3 blockDim(256);
      dim3 gridDim(CEIL_DIV(N, 128), CEIL_DIV(N, 128));

      for (int i = 0; i < warpup_time; ++i) {
        mysgemm_v4<128, 128, 8, 8, 8>
            <<<gridDim, blockDim>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
      }

      hipDeviceSynchronize();
      checkCudaError(hipMemset(d_C_v1, 0, size), "hipMemset d_C_v1 failed");

      checkCudaError(hipEventRecord(start),
                     "cudaEventRecord(start v1) failed");

      for (int i = 0; i < repeat_time; ++i) {
        mysgemm_v4<128, 128, 8, 8, 8>
            <<<gridDim, blockDim>>>(N, N, N, alpha, d_A, d_B, beta, d_C_v1);
      }
      checkCudaError(hipEventRecord(stop), "cudaEventRecord(stop v1) failed");
      checkCudaError(hipEventSynchronize(stop),
                     "hipEventSynchronize v1 failed");
      float v1_time = 0;
      checkCudaError(hipEventElapsedTime(&v1_time, start, stop),
                     "hipEventElapsedTime v1 failed");

      // 拷贝手写 kernel 结果
      checkCudaError(hipMemcpy(C_v1, d_C_v1, size, hipMemcpyDeviceToHost),
                     "hipMemcpy C_v1 failed");
      // 结果比较
      int error_count = 0;
      for (int i = 0; i < N * N && error_count < 10; ++i) {
        if (fabsf(C_cublas[i] - C_v1[i]) > TOL) {
          error_count++;
        }
      }

      float cublas_gflops =
          repeat_time * 2.0f * N * N * N / (cublas_time * 1e6f);  // GFlops
      float v1_gflops =
          repeat_time * 2.0f * N * N * N / (v1_time * 1e6f);  // GFlops
      // 写入CSV
      csv_file << N << "," << cublas_gflops << "," << v1_gflops << ","
               << (error_count == 0 ? "1" : "0") << std::endl;

      // 释放资源
      hipblasDestroy(handle);
      hipEventDestroy(start);
      hipEventDestroy(stop);
      hipFree(d_A);
      hipFree(d_B);
      hipFree(d_C_v1);

      free(A);
      free(B);
      free(C_cublas);
      free(C_v1);

    } catch (...) {
      std::cerr << "Out of memory or error during testing size: " << N
                << std::endl;
      out_of_memory = true;
    }

    if (!out_of_memory) {
      std::cout << "Finished size: " << N << std::endl;
    } else {
      csv_file << N << ",OOM,OOM,0" << std::endl;
    }
  }

  csv_file.close();

  std::cout << "Benchmark completed. Results saved to 'sgemm_benchmark.csv'"
            << std::endl;
  return 0;
}
