#include <hip/hip_runtime.h>

#include <chrono>  // 用于 CPU 计时
#include <iostream>
#include <numeric>
#include <vector>

__inline__ __device__ float block_reduce(float val) {
  const int tid = threadIdx.x;
  const int warpSize = 32;
  int lane = tid % warpSize;
  int warp_id = tid / warpSize;

  // First level: warp reduce using shuffle
#pragma unroll
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down_sync(0xFFFFFFFF, val, offset);

  // Only one warp (warp 0) participates in the second reduction
  __shared__ float warpSums[32];
  if (lane == 0) {
    warpSums[warp_id] = val;  // Store warp sum in shared memory
  }
  __syncthreads();

  if (warp_id == 0) {
    val = (tid < blockDim.x / warpSize) ? warpSums[tid] : 0.0f;
#pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
      val += __shfl_down_sync(0xFFFFFFFF, val, offset);
  }
  return val;
}

__global__ void reduce_v4(const float* in, float* out, int n) {
  float sum = 0.0f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n;
       i += gridDim.x * blockDim.x) {
    sum += in[i];
  }

  sum = block_reduce(sum);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = sum;
  }
}

float reduce_cpu(const std::vector<float>& data) {
  float sum = 0.0f;
  for (float val : data) {
    sum += val;
  }
  return sum;
}

const int BLOCK_SIZE = 1024;
const int N = 1024 * 1024;  // 1M elements
int main() {
  int num_blocks = ((N + BLOCK_SIZE - 1) / BLOCK_SIZE);

  std::vector<float> h_data(N);

  for (int i = 0; i < N; i++) {
    h_data[i] = 1.0f;  // 简单起见，全部初始化为1.0
  }

  // -------------------------------
  // CPU 计时开始
  auto cpu_start = std::chrono::high_resolution_clock::now();

  float cpu_result = reduce_cpu(h_data);

  auto cpu_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
  // CPU 计时结束
  // -------------------------------

  std::cout << "CPU result: " << cpu_result << std::endl;
  std::cout << "CPU time: " << cpu_duration.count() << " ms" << std::endl;

  float *d_data, *d_result;
  float* d_final_result;
  float gpu_result;

  hipMalloc(&d_data, N * sizeof(float));
  hipMalloc(&d_result, num_blocks * sizeof(float));
  hipMalloc(&d_final_result, 1 * sizeof(float));

  hipMemcpy(d_data, h_data.data(), N * sizeof(float), hipMemcpyHostToDevice);

  // -------------------------------
  // GPU 计时开始 (CUDA Events)
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  reduce_v4<<<num_blocks, BLOCK_SIZE>>>(d_data, d_result, N);
  reduce_v4<<<1, num_blocks>>>(d_result, d_final_result, num_blocks);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  // GPU 计时结束
  // -------------------------------

  std::cout << "GPU kernel time: " << milliseconds << " ms" << std::endl;

  hipMemcpy(&gpu_result, d_final_result, sizeof(float),
             hipMemcpyDeviceToHost);
  std::cout << "GPU result: " << gpu_result << std::endl;

  if (abs(cpu_result - gpu_result) < 1e-5) {
    std::cout << "Result verified successfully!" << std::endl;
  } else {
    std::cout << "Result verification failed!" << std::endl;
  }

  // 清理资源
  hipFree(d_data);
  hipFree(d_result);
  hipFree(d_final_result);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}