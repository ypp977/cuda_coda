#include <iostream>
#include <hip/hip_runtime.h>

__global__ void hist(int8_t *input, int *hist, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    for (int idx = i; idx < n; idx += gridDim.x * blockDim.x)
    {
        int8_t in = input[idx];
        if (in >= 0 && in < 256)
        {
            atomicAdd(&hist[in], 1);
        }
    }
}

int main()
{
    int M = 3;
    int N = 3;
    int size = M * N;
    int8_t *input = new int8_t[size];
    input[0] = 1;
    input[1] = 2;
    input[2] = 3;

    input[3] = 2;
    input[4] = 3;
    input[5] = 4;

    input[6] = 3;
    input[7] = 4;
    input[8] = 5;

    int8_t *d_input;
    int *d_hist;
    hipMalloc(&d_input, size * sizeof(int8_t));
    hipMalloc(&d_hist, 256 * sizeof(int));
    hipMemset(d_hist, 0, 256 * sizeof(int));

    dim3 block_size(2);
    dim3 grid_size(2);
    hipMemcpy(d_input, input, sizeof(int8_t) * size, hipMemcpyHostToDevice);
    hist<<<grid_size, block_size>>>(d_input, d_hist, size);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("cuda error:%d\n", err);
    }

    int h_hist[256];
    hipMemcpy(h_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 1; i <= 6; ++i)
    {
        printf("%d : %d\n", i, h_hist[i]);
    }
    delete[] input;
}