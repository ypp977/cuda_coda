#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

int main() {
  int size = 4 * sizeof(int);
  int *h_a = (int *)malloc(size);
  int *h_b = (int *)malloc(size);
  int *d_a;

  for (int i = 0; i < 4; i++) {
    h_a[i] = i;
  }
  hipMalloc((void **)&d_a, size);
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

  hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost);
  for (int i = 0; i < 4; ++i) {
    printf("h_b[%d] = %d\n", i, h_b[i]);
  }

  hipFree(d_a);
  free(h_a);
  free(h_b);
  return 0;
}
