// rmsnorm_cuda_test.cpp
#include <hip/hip_runtime.h>

#include <chrono>
#include <cmath>
#include <iostream>
#include <random>
#include <vector>

void row_rmsnorm_f32_dim_cpu(float* in, float* weight, float* out, int batch,
                             int size, float eps) {
  for (int i = 0; i < batch; ++i) {
    float* in_ptr = in + i * size;
    float* out_ptr = out + i * size;

    float sum = 0.0f;
    for (int j = 0; j < size; ++j) {
      float val = in_ptr[j];
      sum += val * val;
    }
    float rms = 1.0f / std::sqrt(sum / static_cast<float>(size) + eps);

    for (int j = 0; j < size; ++j) {
      float x = in_ptr[j] * weight[j];
      out_ptr[j] = x * rms;
    }
  }
}

__inline__ __device__ float block_reduce(float val) {
  const int tid = threadIdx.x;
  const int warpSize = 32;
  int lane = tid % warpSize;
  int warp_id = tid / warpSize;

  // Warp-level reduction
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down_sync(0xFFFFFFFF, val, offset);

  // Write warp result to shared memory
  __shared__ float warpSums[32];  // Max 32 warps per block
  if (lane == 0) {
    warpSums[warp_id] = val;
  }
  __syncthreads();

  // Final reduction: only first warp participates
  if (warp_id == 0) {
    val = (tid < (blockDim.x + warpSize - 1) / warpSize) ? warpSums[tid] : 0.0f;
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
      val += __shfl_down_sync(0xFFFFFFFF, val, offset);
  } else {
    val = 0.0f;
  }
  return val;
}

__global__ void row_rmsnorm_f32_dim_simd(float* in, float* wei, float* out,
                                         int batch, int size, float eps) {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  if (bid >= batch) {
    return;
  }

  float* block_in = in + bid * size;
  float* block_out = out + bid * size;
  constexpr int pack_size = 4;
  const int pack_num = size / pack_size;
  const int pack_off = pack_size * pack_num;

  float sum = 0.0f;
  float4* in_pack = reinterpret_cast<float4*>(block_in);
  for (int i = tid; i < pack_num; i += blockDim.x) {
    float4 in_float4 = *(in_pack + i);
    sum += in_float4.x * in_float4.x;
    sum += in_float4.y * in_float4.y;
    sum += in_float4.z * in_float4.z;
    sum += in_float4.w * in_float4.w;
  }

  for (int i = pack_off + tid; i < size; i += blockDim.x) {
    sum += block_in[i] * block_in[i];
  }

  __shared__ float shared_val;
  sum = block_reduce(sum);

  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  sum = shared_val;

  const float scale = rsqrtf(sum / static_cast<float>(size) + eps);
  float4* wei_pack = reinterpret_cast<float4*>(wei);
  float4* out_pack = reinterpret_cast<float4*>(block_out);
  for (int i = tid; i < pack_num; i += blockDim.x) {
    float4 in_float4 = *(in_pack + i);
    float4 wei_float4 = *(wei_pack + i);
    *(out_pack + i) = make_float4(
        scale * in_float4.x * wei_float4.x, scale * in_float4.y * wei_float4.y,
        scale * in_float4.z * wei_float4.z, scale * in_float4.w * wei_float4.w);
  }

  for (int i = pack_off + tid; i < size; i += blockDim.x) {
    block_out[i] = wei[i] * block_in[i] * scale;
  }
}

__global__ void row_rmsnorm_f32_dim(float* in, float* wei, float* out,
                                    int batch, int size, float eps) {
  const int bid = blockIdx.x;
  if (bid >= batch) return;

  float* block_in = in + bid * size;
  float* block_out = out + bid * size;
  float sum = 0.0f;

  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    float x = block_in[i];
    sum += x * x;
  }
  __shared__ float shared_val;
  sum = block_reduce(sum);

  if (threadIdx.x == 0) {
    shared_val = sum;
  }
  __syncthreads();
  sum = shared_val;

  const float scale = rsqrtf(sum / static_cast<float>(size) + eps);
  for (int i = threadIdx.x; i < size; i += blockDim.x) {
    float x = block_in[i] * wei[i];
    block_out[i] = x * scale;
  }
}

float compute_max_error(const std::vector<float>& cpu_out,
                        const std::vector<float>& cuda_out, int n) {
  float max_err = 0.0f;
  for (int i = 0; i < n; ++i) {
    float err = std::abs(cpu_out[i] - cuda_out[i]);
    max_err = std::max(max_err, err);
    if (max_err > 1.f) {
      std::cout << "Error at index " << i << ": CPU = " << cpu_out[i]
                << ", CUDA = " << cuda_out[i] << ", Error = " << err << "\n";
      break;
    }
  }
  return max_err;
}

// ----------------------------
// Main Function
// ----------------------------
int main() {
  const int batch = 16;
  const int size = 1024;
  const float eps = 1e-6f;
  const int total = batch * size;

  // Host memory
  std::vector<float> h_input(total);
  std::vector<float> h_weight(size);
  std::vector<float> h_output_cpu(total);
  std::vector<float> h_output_cuda(total);

  // Random init
  std::random_device rd;
  std::mt19937 gen(rd());
  std::normal_distribution<float> dis(0.0f, 1.0f);

  for (int i = 0; i < total; ++i) {
    h_input[i] = dis(gen);
  }
  for (int i = 0; i < size; ++i) {
    h_weight[i] = dis(gen);
  }

  // CPU version
  auto start = std::chrono::high_resolution_clock::now();
  row_rmsnorm_f32_dim_cpu(h_input.data(), h_weight.data(), h_output_cpu.data(),
                          batch, size, eps);
  auto end = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "CPU RMSNorm took " << duration.count() << " microseconds.\n";

  // CUDA setup
  float *d_input, *d_weight, *d_output;
  hipMalloc(&d_input, total * sizeof(float));
  hipMalloc(&d_weight, size * sizeof(float));
  hipMalloc(&d_output, total * sizeof(float));

  hipMemcpy(d_input, h_input.data(), total * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight.data(), size * sizeof(float),
             hipMemcpyHostToDevice);

  // Kernel launch config
  const int block_size = 1024;
  const int grid_size = batch;  // One block per batch row
  dim3 grid(grid_size);
  dim3 block(block_size);

  // CUDA timing with events
  hipEvent_t start_event, stop_event;
  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);

  int warpup = 10;
  for (int i = 0; i < warpup; i++) {
    // Warm-up run
    row_rmsnorm_f32_dim<<<grid, block>>>(d_input, d_weight, d_output, batch,
                                         size, eps);
  }
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != 0) {
    printf("cuda error:%d\n", err);
  }
  hipEventRecord(start_event);
  // row_rmsnorm_f32_dim<<<grid, block>>>(d_input, d_weight, d_output, batch,
  // size, eps);
  int test_iter = 10;
  for (int i = 0; i < test_iter; ++i) {
    row_rmsnorm_f32_dim<<<grid, block>>>(d_input, d_weight, d_output, batch,
                                         size, eps);
  }
  hipEventRecord(stop_event);

  // Wait and measure
  hipEventSynchronize(stop_event);
  float cuda_time;
  hipEventElapsedTime(&cuda_time, start_event, stop_event);  // ms

  // Copy result back
  hipMemcpy(h_output_cuda.data(), d_output, total * sizeof(float),
             hipMemcpyDeviceToHost);

  std::cout << "CUDA RMSNorm took " << cuda_time * 1000 / test_iter
            << " microseconds.\n";

  // Compare results
  float max_error = compute_max_error(h_output_cpu, h_output_cuda, total);
  std::cout << "Max absolute error (CPU vs CUDA): " << max_error << "\n";

  // Optional: print first few values
  std::cout << "\nFirst 10 outputs (CPU vs CUDA):\n";
  for (int i = 0; i < 10; ++i) {
    std::cout << "CPU: " << h_output_cpu[i] << " | CUDA: " << h_output_cuda[i]
              << " | Diff: " << std::abs(h_output_cpu[i] - h_output_cuda[i])
              << "\n";
  }

  // Cleanup
  hipFree(d_input);
  hipFree(d_weight);
  hipFree(d_output);
  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);

  return 0;
}
