#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_world(void)
{
    printf("block idx:%d thread idx: %d\n", blockIdx.x, threadIdx.x);
    if (threadIdx.x == 0)
    {
        printf("GPU thread idx: %d Hello world!\n", threadIdx.x);
    }
}

int main(int argc, char** argv)
{
    printf("CPU: Hello world!\n");
    hello_world<<<1, 10>>>(); // 有20个线程，组成了两个线程块，一个线程块有10个线程。
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        return 1;
    }
    else
    {
        std::cout << "GPU: Hello world finished!" << std::endl;
    }
    std::cout << "CPU: Hello world finished!" << std::endl;
    return 0;
}
